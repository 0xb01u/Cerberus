#include "hip/hip_runtime.h"
/*
 * Simplified simulation of life evolution
 *
 * Computacion Paralela, Grado en Informatica (Universidad de Valladolid)
 * 2019/2020
 *
 * v1.5
 *
 * CHANGES:
 * 1) Float values have been substituted by fixed point arithmetics 
 *	using integers. To simplify, the fixed point arithmetics are done 
 *	with PRECISION in base 10. See precision constant in int_float.h
 * 2) It uses a portable approximation to trigonometric functions using
 *	Taylor polynomials. 
 * 3) nrand48 function has been extracted from glibc source code and 
 *	its internal API simplified to allow its use in the GPU.
 *
 * (c) 2020, Arturo Gonzalez Escribano
 */
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<math.h>
#include<stdbool.h>
#include<cputils.h>
#include<hip/hip_runtime.h>
#include<int_float.h>

/* 
 * Constants: Converted to fixed point with the given PRECISION
 */
#define ENERGY_NEEDED_TO_LIVE		PRECISION / 10	// Equivalent to 0.1
#define ENERGY_NEEDED_TO_MOVE		PRECISION	// Equivalent to 1.0
#define ENERGY_SPENT_TO_LIVE		PRECISION / 5	// Equivalent to 0.2
#define ENERGY_SPENT_TO_MOVE		PRECISION	// Equivalent to 1.0
#define ENERGY_NEEDED_TO_SPLIT		PRECISION * 20	// Equivalent to 20.0


/* Structure to store data of a cell */
typedef struct {
	int pos_row, pos_col;		// Position
	int mov_row, mov_col;		// Direction of movement
	int choose_mov[3];		// Genes: Probabilities of 0 turning-left; 1 advance; 2 turning-right
	int storage;			// Food/Energy stored
	int age;			// Number of steps that the cell has been alive
	unsigned short random_seq[3];	// Status value of its particular random sequence
	bool alive;			// Flag indicating if the cell is still alive
} Cell;


/* Structure for simulation statistics */
typedef struct {
	int history_total_cells;	// Accumulated number of cells created
	int history_dead_cells;		// Accumulated number of dead cells
	int history_max_alive_cells;	// Maximum number of cells alive in a step
	int history_max_new_cells;	// Maximum number of cells created in a step
	int history_max_dead_cells;	// Maximum number of cells died in a step
	int history_max_age;		// Maximum age achieved by a cell
	int history_max_food;		// Maximum food level in a position of the culture
} Statistics;


/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 *
 *	USE THIS SPACE FOR YOUR KERNEL OR DEVICE FUNTIONS
 *
 */

#include "taylor_trig.h"
#include "glibc_nrand48.h"

/*
 * Get an uniformly distributed random number between 0 and max
 * It uses glibc_nrand, that returns a number between 0 and 2^31
 */
#define int_urand48( max, seq )	(int)( (long)(max) * glibc_nrand48( seq ) / 2147483648 )

/* 
 * Macro function to simplify accessing with two coordinates to a flattened array
 * 	This macro-function can be modified by the students if needed
 *
 */
#define accessMat( arr, exp1, exp2 )	arr[ (int)(exp1) * columns + (int)(exp2) ]

/*
 * Function: Choose a new direction of movement for a cell
 * 	This function can be changed and/or optimized by the students
 */
void cell_new_direction( Cell *cell ) {
	int angle = int_urand48( INT_2PI, cell->random_seq );
	cell->mov_row = taylor_sin( angle );
	cell->mov_col = taylor_cos( angle );
}

/*
 * Function: Mutation of the movement genes on a new cell
 * 	This function can be changed and/or optimized by the students
 */
void cell_mutation( Cell *cell ) {
	/* 1. Select which genes change:
	 	0 Left grows taking part of the Advance part
	 	1 Advance grows taking part of the Left part
	 	2 Advance grows taking part of the Right part
	 	3 Right grows taking part of the Advance part
	*/
	int mutation_type = int_urand48( 4, cell->random_seq );
	/* 2. Select the amount of mutation (up to 50%) */
	int mutation_percentage = int_urand48( PRECISION / 2, cell->random_seq );
	/* 3. Apply the mutation */
	int mutation_value;
	switch( mutation_type ) {
		case 0:
			mutation_value = intfloatMult( cell->choose_mov[1] , mutation_percentage );
			cell->choose_mov[1] -= mutation_value;
			cell->choose_mov[0] += mutation_value;
			break;
		case 1:
			mutation_value = intfloatMult( cell->choose_mov[0] , mutation_percentage );
			cell->choose_mov[0] -= mutation_value;
			cell->choose_mov[1] += mutation_value;
			break;
		case 2:
			mutation_value = intfloatMult( cell->choose_mov[2] , mutation_percentage );
			cell->choose_mov[2] -= mutation_value;
			cell->choose_mov[1] += mutation_value;
			break;
		case 3:
			mutation_value = intfloatMult( cell->choose_mov[1] , mutation_percentage );
			cell->choose_mov[1] -= mutation_value;
			cell->choose_mov[2] += mutation_value;
			break;
	}
	/* 4. Correct potential precision problems */
	cell->choose_mov[2] = PRECISION - cell->choose_mov[1] - cell->choose_mov[0];
}

/*
 * CUDA block reduction
 * Inputs: 
 *	Device pointer to an array of int of any size
 *	Size of the array
 *	Device pointer to an int to store the result
 * 
 * Launching parameters:
 *	One-dimesional grid of any size
 *	Any valid block size
 *	Dynamic shared memory size equal to: sizeof(int) * block size
 *
 * (c) 2020, Arturo Gonzalez-Escribano
 * Simplification for an assignment in a Parallel Computing course,
 * Computing Engineering Degree, Universidad de Valladolid
 * Academic year 2019/2020
 */
__global__ void reductionMax(int* array, int size, int *result)
{
	int globalPos = threadIdx.x + blockIdx.x * blockDim.x;

	extern __shared__ int buffer[ ];
	if ( globalPos < size ) { 
		buffer[ threadIdx.x ] = array[ globalPos ];
	}
	else buffer[ threadIdx.x ] = 0.0f;
	__syncthreads();

	for( int step=blockDim.x/2; step>=1; step /= 2 ) {
		if ( threadIdx.x < step )
			if ( buffer[ threadIdx.x ] < buffer[ threadIdx.x + step ] )
				buffer[ threadIdx.x ] = buffer[ threadIdx.x + step ];
		if ( step > 32 )
			__syncthreads();
	}

	if ( threadIdx.x == 0 )
		atomicMax( result, buffer[0] );
}


/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */


#ifdef DEBUG
/* 
 * Function: Print the current state of the simulation 
 */
void print_status( int iteration, int rows, int columns, int *culture, int num_cells, Cell *cells, int num_cells_alive, Statistics sim_stat ) {
	/* 
	 * You don't need to optimize this function, it is only for pretty printing and debugging purposes.
	 * It is not compiled in the production versions of the program.
	 * Thus, it is never used when measuring times in the leaderboard
	 */
	int i,j;

	printf("Iteration: %d\n", iteration );
	printf("+");
	for( j=0; j<columns; j++ ) printf("---");
	printf("+\n");
	for( i=0; i<rows; i++ ) {
		printf("|");
		for( j=0; j<columns; j++ ) {
			char symbol;
			if ( accessMat( culture, i, j ) >= 20 * PRECISION ) symbol = '+';
			else if ( accessMat( culture, i, j ) >= 10 * PRECISION ) symbol = '*';
			else if ( accessMat( culture, i, j ) >= 5 * PRECISION ) symbol = '.';
			else symbol = ' ';

			int t;
			int counter = 0;
			for( t=0; t<num_cells; t++ ) {
				int row = (int)(cells[t].pos_row / PRECISION);
				int col = (int)(cells[t].pos_col / PRECISION);
				if ( cells[t].alive && row == i && col == j ) {
					counter ++;
				}
			}
			if ( counter > 9 ) printf("(M)" );
			else if ( counter > 0 ) printf("(%1d)", counter );
			else printf(" %c ", symbol );
		}
		printf("|\n");
	}
	printf("+");
	for( j=0; j<columns; j++ ) printf("---");
	printf("+\n");
	printf("Num_cells_alive: %04d\nHistory( Cells: %04d, Dead: %04d, Max.alive: %04d, Max.new: %04d, Max.dead: %04d, Max.age: %04d, Max.food: %6f )\n\n", 
		num_cells_alive, 
		sim_stat.history_total_cells, 
		sim_stat.history_dead_cells, 
		sim_stat.history_max_alive_cells, 
		sim_stat.history_max_new_cells, 
		sim_stat.history_max_dead_cells, 
		sim_stat.history_max_age,
		(float)sim_stat.history_max_food / PRECISION
	);
}
#endif

/*
 * Function: Print usage line in stderr
 */
void show_usage( char *program_name ) {
	fprintf(stderr,"Usage: %s ", program_name );
	fprintf(stderr,"<rows> <columns> <maxIter> <max_food> <food_density> <food_level> <short_rnd1> <short_rnd2> <short_rnd3> <num_cells>\n");
	fprintf(stderr,"\tOptional arguments for special food spot: [ <row> <col> <size_rows> <size_cols> <density> <level> ]\n");
	fprintf(stderr,"\n");
}


/*
 * MAIN PROGRAM
 */
int main(int argc, char *argv[]) {
	int i,j;

	// Simulation data
	int max_iter;			// Maximum number of simulation steps
	int rows, columns;		// Cultivation area sizes
	int *culture;			// Cultivation area values
	int *culture_cells;		// Ancillary structure to count the number of cells in a culture space

	float max_food;			// Maximum level of food on any position
	float food_density;		// Number of food sources introduced per step
	float food_level;		// Maximum number of food level in a new source

	bool food_spot_active = false;	// Special food spot: Active
	int food_spot_row = 0;		// Special food spot: Initial row
	int food_spot_col = 0;		// Special food spot: Initial row
	int food_spot_size_rows = 0;	// Special food spot: Rows size
	int food_spot_size_cols = 0;	// Special food spot: Cols size
	float food_spot_density = 0.0f;	// Special food spot: Food density
	float food_spot_level = 0.0f;	// Special food spot: Food level

	unsigned short init_random_seq[3];	// Status of the init random sequence
	unsigned short food_random_seq[3];	// Status of the food random sequence
	unsigned short food_spot_random_seq[3];	// Status of the special food spot random sequence

	int	num_cells;		// Number of cells currently stored in the list
	Cell	*cells;			// List to store cells information

	// Statistics
	Statistics sim_stat;	
	sim_stat.history_total_cells = 0;
	sim_stat.history_dead_cells = 0;
	sim_stat.history_max_alive_cells = 0;
	sim_stat.history_max_new_cells = 0;
	sim_stat.history_max_dead_cells = 0;
	sim_stat.history_max_age = 0;
	sim_stat.history_max_food = 0.0f;

	/* 1. Read simulation arguments */
	/* 1.1. Check minimum number of arguments */
	if (argc < 11) {
		fprintf(stderr, "-- Error: Not enough arguments when reading configuration from the command line\n\n");
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}

	/* 1.2. Read culture sizes, maximum number of iterations */
	rows = atoi( argv[1] );
	columns = atoi( argv[2] );
	max_iter = atoi( argv[3] );

	/* 1.3. Food data */
	max_food = atof( argv[4] );
	food_density = atof( argv[5] );
	food_level = atof( argv[6] );

	/* 1.4. Read random sequences initializer */
	for( i=0; i<3; i++ ) {
		init_random_seq[i] = (unsigned short)atoi( argv[7+i] );
	}

	/* 1.5. Read number of cells */
	num_cells = atoi( argv[10] );

	/* 1.6. Read special food spot */
	if (argc > 11 ) {
		if ( argc < 17 ) {
			fprintf(stderr, "-- Error in number of special-food-spot arguments in the command line\n\n");
			show_usage( argv[0] );
			exit( EXIT_FAILURE );
		}
		else {
			food_spot_active = true;
			food_spot_row = atoi( argv[11] );
			food_spot_col = atoi( argv[12] );
			food_spot_size_rows = atoi( argv[13] );
			food_spot_size_cols = atoi( argv[14] );
			food_spot_density = atof( argv[15] );
			food_spot_level = atof( argv[16] );

			// Check non-used trailing arguments
			if ( argc > 17 ) {
				fprintf(stderr, "-- Error: too many arguments in the command line\n\n");
				show_usage( argv[0] );
				exit( EXIT_FAILURE );
			}
		}
	}

#ifdef DEBUG
	/* 1.7. Print arguments */
	printf("Arguments, Rows: %d, Columns: %d, max_iter: %d\n", rows, columns, max_iter);
	printf("Arguments, Max.food: %f, Food density: %f, Food level: %f\n", max_food, food_density, food_level);
	printf("Arguments, Init Random Sequence: %hu,%hu,%hu\n", init_random_seq[0], init_random_seq[1], init_random_seq[2]);
	if ( food_spot_active ) {
		printf("Arguments, Food_spot, pos(%d,%d), size(%d,%d), Density: %f, Level: %f\n",
			food_spot_row, food_spot_col, food_spot_size_rows, food_spot_size_cols, food_spot_density, food_spot_level );
	}
	printf("Initial cells: %d\n", num_cells );
#endif // DEBUG


	/* 1.8. Initialize random sequences for food dropping */
	for( i=0; i<3; i++ ) {
		food_random_seq[i] = (unsigned short)glibc_nrand48( init_random_seq );
		food_spot_random_seq[i] = (unsigned short)glibc_nrand48( init_random_seq );
	}

	/* 1.9. Initialize random sequences of cells */
	cells = (Cell *)malloc( sizeof(Cell) * (size_t)num_cells );
	if ( cells == NULL ) {
		fprintf(stderr,"-- Error allocating: %d cells\n", num_cells );
		exit( EXIT_FAILURE );
	}
	for( i=0; i<num_cells; i++ ) {
		// Initialize the cell ramdom sequences
		for( j=0; j<3; j++ ) 
			cells[i].random_seq[j] = (unsigned short)glibc_nrand48( init_random_seq );
	}


#ifdef DEBUG
	/* 1.10. Print random seed of the initial cells */
	/*
	printf("Initial cells random seeds: %d\n", num_cells );
	for( i=0; i<num_cells; i++ )
		printf("\tCell %d, Random seq: %hu,%hu,%hu\n", i, cells[i].random_seq[0], cells[i].random_seq[1], cells[i].random_seq[2] );
	*/
#endif // DEBUG


	// CUDA start
	hipSetDevice(0);
	hipDeviceSynchronize();

	/* 2. Start global timer */
	double ttotal = cp_Wtime();

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 *
 */

#include "cuda_check.h"
#include "cuda_time.h"

	/* 3. Initialize culture surface and initial cells */
	culture = (int *)malloc( sizeof(int) * (size_t)rows * (size_t)columns );
	culture_cells = (int *)malloc( sizeof(int) * (size_t)rows * (size_t)columns );
#ifdef DEVELOPMENT
	if ( culture == NULL || culture_cells == NULL ) {
		fprintf(stderr,"-- Error allocating culture structures for size: %d x %d \n", rows, columns );
		exit( EXIT_FAILURE );
	}
#endif // DEVELOPMENT
	// 3.1
	time_start();
	for( i=0; i<rows; i++ )
		for( j=0; j<columns; j++ ) 
			accessMat( culture, i, j ) = 0;
	time_end(time3_1);

	// 3.2
	time_start();
	for( i=0; i<num_cells; i++ ) {
		cells[i].alive = true;
		// Initial age: Between 1 and 20 
		cells[i].age = 1 + int_urand48( 19, cells[i].random_seq );
		// Initial storage: Between 10 and 20 units
		cells[i].storage = 10 * PRECISION + int_urand48( 10 * PRECISION, cells[i].random_seq );
		// Initial position: Anywhere in the culture arena
		cells[i].pos_row = int_urand48( rows * PRECISION, cells[i].random_seq );
		cells[i].pos_col = int_urand48( columns * PRECISION, cells[i].random_seq );
		// Movement direction: Unity vector in a random direction
		cell_new_direction( &cells[i] );
		// Movement genes: Probabilities of advancing or changing direction: The sum should be 1.00
		cells[i].choose_mov[0] = PRECISION / 3;
		cells[i].choose_mov[2] = PRECISION / 3;
		cells[i].choose_mov[1] = PRECISION - cells[i].choose_mov[0] - cells[i].choose_mov[2];
	}

	// Statistics: Initialize total number of cells, and max. alive
	sim_stat.history_total_cells = num_cells;
	sim_stat.history_max_alive_cells = num_cells;
	time_end(time3_2);

#ifdef DEBUG
	/* Show initial cells data */
	printf("Initial cells data: %d\n", num_cells );
	for( i=0; i<num_cells; i++ ) {
		printf("\tCell %d, Pos(%f,%f), Mov(%f,%f), Choose_mov(%f,%f,%f), Storage: %f, Age: %d\n",
				i, 
				(float)cells[i].pos_row / PRECISION, 
				(float)cells[i].pos_col / PRECISION, 
				(float)cells[i].mov_row / PRECISION, 
				(float)cells[i].mov_col / PRECISION, 
				(float)cells[i].choose_mov[0] / PRECISION, 
				(float)cells[i].choose_mov[1] / PRECISION, 
				(float)cells[i].choose_mov[2] / PRECISION, 
				(float)cells[i].storage / PRECISION,
				cells[i].age );
	}
#endif // DEBUG

	/* 4. Simulation */
	int current_max_food = 0;
	int num_cells_alive = num_cells;
	int iter;
	int max_food_int = max_food * PRECISION;

	int num_new_sources = (int)(rows * columns * food_density);
	int num_new_sources_spot = food_spot_active ? (int)(food_spot_size_rows * food_spot_size_cols * food_spot_density) : 0;
	for( iter=0; iter<max_iter && current_max_food <= max_food_int && num_cells_alive > 0; iter++ ) {
		update_times();

		int step_new_cells = 0;
		int step_dead_cells = 0;

		/* 4.1. Spreading new food */
		time_start();
		// Across the whole culture
		for (i=0; i<num_new_sources; i++) {
			int row = int_urand48( rows, food_random_seq );
			int col = int_urand48( columns, food_random_seq );
			int food = int_urand48( food_level * PRECISION, food_random_seq );
			accessMat( culture, row, col ) = accessMat( culture, row, col ) + food;
		}
		// In the special food spot
		if ( food_spot_active ) {
			for (i=0; i<num_new_sources_spot; i++) {
				int row = food_spot_row + int_urand48( food_spot_size_rows, food_spot_random_seq );
				int col = food_spot_col + int_urand48( food_spot_size_cols, food_spot_random_seq );
				int food = int_urand48( food_spot_level * PRECISION, food_spot_random_seq );
				accessMat( culture, row, col ) = accessMat( culture, row, col ) + food;
			}
		}
		time_end(time4_1);

		/* 4.2. Prepare ancillary data structures */
		time_start();
		/* 4.2.1. Clear ancillary structure of the culture to account alive cells in a position after movement */
		for( i=0; i<rows; i++ )
			for( j=0; j<columns; j++ ) 
				accessMat( culture_cells, i, j ) = 0;
 		/* 4.2.2. Allocate ancillary structure to store the food level to be shared by cells in the same culture place */
		int *food_to_share = (int *)malloc( sizeof(int) * num_cells );
#ifdef DEVELOPMENT
		if ( food_to_share == NULL ) {
			fprintf(stderr,"-- Error allocating food_to_share structures for size: %d x %d \n", rows, columns );
			exit( EXIT_FAILURE );
		}
#endif // DEVELOPMENT
		time_end(time4_2);

		/* 4.3. Cell movements */
		time_start();
		for (i=0; i<num_cells; i++) {
			if ( cells[i].alive ) {
				cells[i].age ++;
				// Statistics: Max age of a cell in the simulation history
				if ( cells[i].age > sim_stat.history_max_age ) sim_stat.history_max_age = cells[i].age;

				/* 4.3.1. Check if the cell has the needed energy to move or keep alive */
				if ( cells[i].storage < ENERGY_NEEDED_TO_LIVE ) {
					// Cell has died
					cells[i].alive = false;
					num_cells_alive --;
					step_dead_cells ++;
					continue;
				}
				if ( cells[i].storage < ENERGY_NEEDED_TO_MOVE ) {
					// Almost dying cell, it cannot move, only if enough food is dropped here it will survive
					cells[i].storage -= ENERGY_SPENT_TO_LIVE;
				}
				else {
					// Consume energy to move
					cells[i].storage -= ENERGY_SPENT_TO_MOVE;
						
					/* 4.3.2. Choose movement direction */
					int prob = int_urand48( PRECISION, cells[i].random_seq );
					if ( prob < cells[i].choose_mov[0] ) {
						// Turn left (90 degrees)
						int tmp = cells[i].mov_col;
						cells[i].mov_col = cells[i].mov_row;
						cells[i].mov_row = -tmp;
					}
					else if ( prob >= cells[i].choose_mov[0] + cells[i].choose_mov[1] ) {
						// Turn right (90 degrees)
						int tmp = cells[i].mov_row;
						cells[i].mov_row = cells[i].mov_col;
						cells[i].mov_col = -tmp;
					}
					// else do not change the direction
					
					/* 4.3.3. Update position moving in the choosen direction*/
					cells[i].pos_row += cells[i].mov_row;
					cells[i].pos_col += cells[i].mov_col;
					// Periodic arena: Left/Rigth edges are connected, Top/Bottom edges are connected
					if ( cells[i].pos_row < 0 ) cells[i].pos_row += rows * PRECISION;
					if ( cells[i].pos_row >= rows * PRECISION) cells[i].pos_row -= rows * PRECISION;
					if ( cells[i].pos_col < 0 ) cells[i].pos_col += columns * PRECISION;
					if ( cells[i].pos_col >= columns * PRECISION) cells[i].pos_col -= columns * PRECISION;
				}

				/* 4.3.4. Annotate that there is one more cell in this culture position */
				accessMat( culture_cells, cells[i].pos_row / PRECISION, cells[i].pos_col / PRECISION ) += 1;
				/* 4.3.5. Annotate the amount of food to be shared in this culture position */
				food_to_share[i] = accessMat( culture, cells[i].pos_row / PRECISION, cells[i].pos_col / PRECISION );
			}
		} // End cell movements
		time_end(time4_3);
		
		/* 4.4. Cell actions */
		time_start();
		// Space for the list of new cells (maximum number of new cells is num_cells)
		Cell *new_cells = (Cell *)malloc( sizeof(Cell) * num_cells );
#ifdef DEVELOPMENT
		if ( new_cells == NULL ) {
			fprintf(stderr,"-- Error allocating new cells structures for: %d cells\n", num_cells );
			exit( EXIT_FAILURE );
		}
#endif // DEVELOPMENT

		for (i=0; i<num_cells; i++) {
			if ( cells[i].alive ) {
				/* 4.4.1. Food harvesting */
				int food = food_to_share[i];
				int count = accessMat( culture_cells, cells[i].pos_row / PRECISION, cells[i].pos_col / PRECISION );
				int my_food = food / count;
				cells[i].storage += my_food;

				/* 4.4.2. Split cell if the conditions are met: Enough maturity and energy */
				if ( cells[i].age > 30 && cells[i].storage > ENERGY_NEEDED_TO_SPLIT ) {
					// Split: Create new cell
					num_cells_alive ++;
					sim_stat.history_total_cells ++;
					step_new_cells ++;

					// New cell is a copy of parent cell
					new_cells[ step_new_cells-1 ] = cells[i];

					// Split energy stored and update age in both cells
					cells[i].storage /= 2;
					new_cells[ step_new_cells-1 ].storage /= 2;
					cells[i].age = 1;
					new_cells[ step_new_cells-1 ].age = 1;

					// Random seed for the new cell, obtained using the parent random sequence
					new_cells[ step_new_cells-1 ].random_seq[0] = (unsigned short)glibc_nrand48( cells[i].random_seq );
					new_cells[ step_new_cells-1 ].random_seq[1] = (unsigned short)glibc_nrand48( cells[i].random_seq );
					new_cells[ step_new_cells-1 ].random_seq[2] = (unsigned short)glibc_nrand48( cells[i].random_seq );

					// Both cells start in random directions
					cell_new_direction( &cells[i] );
					cell_new_direction( &new_cells[ step_new_cells-1 ] );
				
					// Mutations of the movement genes in both cells
					cell_mutation( &cells[i] );
					cell_mutation( &new_cells[ step_new_cells-1 ] );
				}
			}
		} // End cell actions
		time_end(time4_4);

		/* 4.5. Clean ancillary data structures */
		time_start();
		/* 4.5.1. Clean the food consumed by the cells in the culture data structure */
		for (i=0; i<num_cells; i++) {
			if ( cells[i].alive ) {
				accessMat( culture, cells[i].pos_row / PRECISION, cells[i].pos_col / PRECISION ) = 0;
			}
		}
		/* 4.5.2. Free the ancillary data structure to store the food to be shared */
		free( food_to_share );
		time_end(time4_5);

		/* 4.6. Clean dead cells from the original list */
		time_start();
		// 4.6.1. Move alive cells to the left to substitute dead cells
		int free_position = 0;
		int alive_in_main_list = 0;
		for( i=0; i<num_cells; i++ ) {
			if ( cells[i].alive ) {
				alive_in_main_list ++;
				if ( free_position != i ) {
					cells[free_position] = cells[i];
				}
				free_position ++;
			}
		}
		// 4.6.2. Reduce the storage space of the list to the current number of cells
		num_cells = alive_in_main_list;
		cells = (Cell *)realloc( cells, sizeof(Cell) * num_cells );
		time_end(time4_6);

		/* 4.7. Join cell lists: Old and new cells list */
		time_start();
		if ( step_new_cells > 0 ) {
			cells = (Cell *)realloc( cells, sizeof(Cell) * ( num_cells + step_new_cells ) );
			for (j=0; j<step_new_cells; j++)
				cells[ num_cells + j ] = new_cells[ j ];
			num_cells += step_new_cells;
		}
		free( new_cells );
		time_end(time4_7);

		/* 4.8. Decrease non-harvested food */
		time_start();
		current_max_food = 0;
		for( i=0; i<rows; i++ )
			for( j=0; j<columns; j++ ) {
				accessMat( culture, i, j ) -= accessMat( culture, i, j ) / 20;
				if ( accessMat( culture, i, j ) > current_max_food ) 
					current_max_food = accessMat( culture, i, j );
			}
		time_end(time4_8);

		/* 4.9. Statistics */
		time_start();
		// Statistics: Max food
		if ( current_max_food > sim_stat.history_max_food ) sim_stat.history_max_food = current_max_food;
		// Statistics: Max new cells per step
		if ( step_new_cells > sim_stat.history_max_new_cells ) sim_stat.history_max_new_cells = step_new_cells;
		// Statistics: Accumulated dead and Max dead cells per step
		sim_stat.history_dead_cells += step_dead_cells;
		if ( step_dead_cells > sim_stat.history_max_dead_cells ) sim_stat.history_max_dead_cells = step_dead_cells;
		// Statistics: Max alive cells per step
		if ( num_cells_alive > sim_stat.history_max_alive_cells ) sim_stat.history_max_alive_cells = num_cells_alive;
		time_end(time4_9);


#ifdef DEBUG
		/* 4.10. DEBUG: Print the current state of the simulation at the end of each iteration */
		print_status( iter, rows, columns, culture, num_cells, cells, num_cells_alive, sim_stat );
#endif // DEBUG
	}

	print_times();
	
/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

	// CUDA stop
	hipDeviceSynchronize();

	/* 5. Stop global time */
	ttotal = cp_Wtime() - ttotal;

#ifdef DEBUG
	printf("List of cells at the end of the simulation: %d\n\n", num_cells );
	for( i=0; i<num_cells; i++ ) {
		printf("Cell %d, Alive: %d, Pos(%f,%f), Mov(%f,%f), Choose_mov(%f,%f,%f), Storage: %f, Age: %d\n",
				i,
				cells[i].alive,
				(float)cells[i].pos_row / PRECISION, 
				(float)cells[i].pos_col / PRECISION, 
				(float)cells[i].mov_row / PRECISION, 
				(float)cells[i].mov_col / PRECISION, 
				(float)cells[i].choose_mov[0] / PRECISION, 
				(float)cells[i].choose_mov[1] / PRECISION, 
				(float)cells[i].choose_mov[2] / PRECISION, 
				(float)cells[i].storage / PRECISION,
				cells[i].age );
	}
#endif // DEBUG

	/* 6. Output for leaderboard */
	printf("\n");
	/* 6.1. Total computation time */
	printf("Time: %lf\n", ttotal );

	/* 6.2. Results: Number of iterations and other statistics */
	printf("Result: %d, ", iter);
	printf("%d, %d, %d, %d, %d, %d, %d, %f\n", 
		num_cells_alive, 
		sim_stat.history_total_cells, 
		sim_stat.history_dead_cells, 
		sim_stat.history_max_alive_cells, 
		sim_stat.history_max_new_cells, 
		sim_stat.history_max_dead_cells, 
		sim_stat.history_max_age,
		(float)sim_stat.history_max_food / PRECISION
	);

	/* 7. Free resources */	
	free( culture );
	free( culture_cells );
	free( cells );

	/* 8. End */
	return 0;
}
